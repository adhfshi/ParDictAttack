#include <stdio.h>
#include <fstream>
//#include <iostream>
#include <string.h>
//#include <vector>
#include <stdlib.h>
//#include <unistd.h>
//#include <time.h>

#include <hip/hip_runtime.h>
//#include <mpi.h>

#define uchar unsigned char // 8-bit byte
#define uint unsigned int // 32-bit word

//define for sha256
#define DBL_INT_ADD(a,b,c) if (a > 0xffffffff - (c)) ++b; a += c;
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }


typedef struct {
	uchar data[64];
	uint datalen;
	uint bitlen[2];
	uint state[8];
} SHA256_CTX;

__constant__ uint k[64] = {
   0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
   0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
   0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
   0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
   0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
   0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
   0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
   0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__constant__ uchar answer[32];


//==============cuda kernel=====================================
__device__ void sha256_transform(SHA256_CTX *ctx, uchar *data){

   uint a,b,c,d,e,f,g,h,i,j,t1,t2,m[64];
      
   for (i=0,j=0; i < 16; ++i, j += 4)
      m[i] = (data[j] << 24) | (data[j+1] << 16) | (data[j+2] << 8) | (data[j+3]);
   for ( ; i < 64; ++i)
      m[i] = SIG1(m[i-2]) + m[i-7] + SIG0(m[i-15]) + m[i-16];

   a = ctx->state[0];
   b = ctx->state[1];
   c = ctx->state[2];
   d = ctx->state[3];
   e = ctx->state[4];
   f = ctx->state[5];
   g = ctx->state[6];
   h = ctx->state[7];
   
   for (i = 0; i < 64; ++i) {
      t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
      t2 = EP0(a) + MAJ(a,b,c);
      h = g;
      g = f;
      f = e;
      e = d + t1;
      d = c;
      c = b;
      b = a;
      a = t1 + t2;
   }
   
   ctx->state[0] += a;
   ctx->state[1] += b;
   ctx->state[2] += c;
   ctx->state[3] += d;
   ctx->state[4] += e;
   ctx->state[5] += f;
   ctx->state[6] += g;
   ctx->state[7] += h;

}

__device__ void sha256(SHA256_CTX *ctx, uchar *data, uchar *hash, int len){
		//init sha256 data structure
		ctx->datalen = 0;
   		ctx->bitlen[0] = 0;
   		ctx->bitlen[1] = 0;
   		ctx->state[0] = 0x6a09e667;
   		ctx->state[1] = 0xbb67ae85;
	   	ctx->state[2] = 0x3c6ef372;
   		ctx->state[3] = 0xa54ff53a;
	   	ctx->state[4] = 0x510e527f;
	   	ctx->state[5] = 0x9b05688c;
	   	ctx->state[6] = 0x1f83d9ab;
	   	ctx->state[7] = 0x5be0cd19;

		//update
		uint i;
		//uint len = 5; //need to fix!!
   		for (i=0; i < len; ++i) { 
    	  	ctx->data[ctx->datalen] = data[i]; 
      		ctx->datalen++; 
      		if (ctx->datalen == 64) { 
         		sha256_transform(ctx,ctx->data);
         		DBL_INT_ADD(ctx->bitlen[0],ctx->bitlen[1],512); 
         		ctx->datalen = 0;
 
      		}  
   		}

		//final
		i = ctx->datalen;

	   	if (ctx->datalen < 56) { 
    	  	ctx->data[i++] = 0x80; 
      		while (i < 56) 
         		ctx->data[i++] = 0x00; 
   		}	
   		else { 
      		ctx->data[i++] = 0x80; 
      		while (i < 64) 
         		ctx->data[i++] = 0x00; 
      	sha256_transform(ctx,ctx->data);
      	memset(ctx->data,0,56); 
   		}
		
		//par here
   		DBL_INT_ADD(ctx->bitlen[0],ctx->bitlen[1],ctx->datalen * 8);
   		ctx->data[63] = ctx->bitlen[0]; 
   		ctx->data[62] = ctx->bitlen[0] >> 8; 
   		ctx->data[61] = ctx->bitlen[0] >> 16; 
   		ctx->data[60] = ctx->bitlen[0] >> 24; 
   		ctx->data[59] = ctx->bitlen[1]; 
   		ctx->data[58] = ctx->bitlen[1] >> 8; 
   		ctx->data[57] = ctx->bitlen[1] >> 16;  
   		ctx->data[56] = ctx->bitlen[1] >> 24; 
   		sha256_transform(ctx,ctx->data);	

		//we can paralized at here
   		for (i=0; i < 4; ++i) { 
      		hash[i]    = (ctx->state[0] >> (24-i*8)) & 0x000000ff; 
      		hash[i+4]  = (ctx->state[1] >> (24-i*8)) & 0x000000ff; 
      		hash[i+8]  = (ctx->state[2] >> (24-i*8)) & 0x000000ff;
      		hash[i+12] = (ctx->state[3] >> (24-i*8)) & 0x000000ff;
      		hash[i+16] = (ctx->state[4] >> (24-i*8)) & 0x000000ff;
      		hash[i+20] = (ctx->state[5] >> (24-i*8)) & 0x000000ff;
      		hash[i+24] = (ctx->state[6] >> (24-i*8)) & 0x000000ff;
      		hash[i+28] = (ctx->state[7] >> (24-i*8)) & 0x000000ff;
   		}	

}

__device__ void sha256_hash(SHA256_CTX *ctx, uchar *data, uchar *hash, int len, int round){
	sha256(ctx, data, hash, len);
	while(round > 1){
		sha256(ctx,hash,hash,32);
		round --;
	}

}


__device__ bool my_strcmp(uchar *str_a, uchar *str_b, uint len){
	for(int i=0; i<len; i++){
		if(str_a[i] != str_b[i])
			return false;
	}
	
	return true;
}


__global__ void sha256_wrap(uchar *pwarray, uchar *target, int* pwcount, uchar *result){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	uchar* data = (uchar*)malloc(pwcount[idx]*sizeof(uchar));

	SHA256_CTX ctx;// = new SHA256_CTX;	
	uchar hash[32];
	int round = 10000, count = 0;	

	for(int i=0; i<idx; i++){
		count += pwcount[i];
	}
	memcpy(data,&pwarray[count],pwcount[idx]*sizeof(uchar));

	sha256_hash(&ctx,data,hash,pwcount[idx],round);

	for (int i=0; i<5;i++){
		if(my_strcmp(hash,&target[32*i],32)){
			memcpy(result,data,pwcount[0]*sizeof(uchar));
		}	
	}
}


//====================================================================
void print_hash(unsigned char hash[]){

   int idx;
   for (idx=0; idx < 32; idx++)
      printf("%02x",hash[idx]);
   printf("\n");
}

extern "C" void hash_pairing(uchar *pwarray, uchar *target, int *pwcount, int num){

	uchar *dev_pwarray, *dev_target, *dev_result;
	uchar *result = new uchar[32]; 
	int *dev_pwcount;

	hipMalloc((void**)&dev_pwarray,strlen((const char*)pwarray));
	hipMemcpy((void*)dev_pwarray, pwarray, strlen((const char*)pwarray),hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_target,160*sizeof(uchar));
	hipMemcpy((void*)dev_target, target, 160*sizeof(uchar),hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_result, 32*sizeof(uchar));
//	cudaMemcpy((void*)dev_result, result, 32*sizeof(uchar), cudaMemcpyHostToDevice);

	hipMalloc((void**)&dev_pwcount,num*sizeof(int));
	hipMemcpy((void*)dev_pwcount,pwcount,num*sizeof(int),hipMemcpyHostToDevice);


	dim3 DimBlock(1024,1);
	dim3 DimGrid(55,1);
	sha256_wrap <<< DimGrid, DimBlock >>> (dev_pwarray, dev_target, dev_pwcount, dev_result);
	hipDeviceSynchronize();
	
	hipMemcpy((void*)result, dev_result, 32*sizeof(uchar), hipMemcpyDeviceToHost);
	if(strlen((const char*)result)!=0)
		printf("password: %s ", result);

	memset(result,0,strlen((const char*) result));
	hipDeviceReset();
	//cudaFree(dev_pwarray); cudaFree(dev_target); cudaFree(dev_result);
}

