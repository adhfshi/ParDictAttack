#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <string.h>
#include <vector>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <hip/hip_runtime.h>
//#include "sha256.h"

#define uchar unsigned char // 8-bit byte
#define uint unsigned int // 32-bit word

//define for sha256
#define DBL_INT_ADD(a,b,c) if (a > 0xffffffff - (c)) ++b; a += c;
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

#define GPUerrchk(ans) { GPUassert((ans), __FILE__, __LINE__); }

typedef struct {
	uchar data[64];
	uint datalen;
	uint bitlen[2];
	uint state[8];
} SHA256_CTX;

__constant__ uint k[64] = {
   0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
   0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
   0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
   0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
   0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
   0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
   0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
   0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__constant__ uchar answer[32];


//==============cuda kernel=====================================
__device__ void sha256_transform(SHA256_CTX *ctx, uchar *data){

   uint a,b,c,d,e,f,g,h,i,j,t1,t2,m[64];
      
   for (i=0,j=0; i < 16; ++i, j += 4)
      m[i] = (data[j] << 24) | (data[j+1] << 16) | (data[j+2] << 8) | (data[j+3]);
   for ( ; i < 64; ++i)
      m[i] = SIG1(m[i-2]) + m[i-7] + SIG0(m[i-15]) + m[i-16];

   a = ctx->state[0];
   b = ctx->state[1];
   c = ctx->state[2];
   d = ctx->state[3];
   e = ctx->state[4];
   f = ctx->state[5];
   g = ctx->state[6];
   h = ctx->state[7];
   
   for (i = 0; i < 64; ++i) {
      t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
      t2 = EP0(a) + MAJ(a,b,c);
      h = g;
      g = f;
      f = e;
      e = d + t1;
      d = c;
      c = b;
      b = a;
      a = t1 + t2;
   }
   
   ctx->state[0] += a;
   ctx->state[1] += b;
   ctx->state[2] += c;
   ctx->state[3] += d;
   ctx->state[4] += e;
   ctx->state[5] += f;
   ctx->state[6] += g;
   ctx->state[7] += h;

}

__device__ void sha256_hash(SHA256_CTX *ctx, uchar *data, uchar *hash, int len, int tround){
	int round = 1;
	while (round > 0){
		//init sha256 data structure
		ctx->datalen = 0;
   		ctx->bitlen[0] = 0;
   		ctx->bitlen[1] = 0;
   		ctx->state[0] = 0x6a09e667;
   		ctx->state[1] = 0xbb67ae85;
	   	ctx->state[2] = 0x3c6ef372;
   		ctx->state[3] = 0xa54ff53a;
	   	ctx->state[4] = 0x510e527f;
	   	ctx->state[5] = 0x9b05688c;
	   	ctx->state[6] = 0x1f83d9ab;
	   	ctx->state[7] = 0x5be0cd19;

		//update
		uint i;
		//uint len = 5; //need to fix!!
   		for (i=0; i < len; ++i) { 
    	  	ctx->data[ctx->datalen] = data[i]; 
      		ctx->datalen++; 
      		if (ctx->datalen == 64) { 
         		sha256_transform(ctx,ctx->data);
         		DBL_INT_ADD(ctx->bitlen[0],ctx->bitlen[1],512); 
         		ctx->datalen = 0;
 
      		}  
   		}

		//final
		i = ctx->datalen;

	   	if (ctx->datalen < 56) { 
    	  	ctx->data[i++] = 0x80; 
      		while (i < 56) 
         		ctx->data[i++] = 0x00; 
   		}	
   		else { 
      		ctx->data[i++] = 0x80; 
      		while (i < 64) 
         		ctx->data[i++] = 0x00; 
      	sha256_transform(ctx,ctx->data);
      	memset(ctx->data,0,56); 
   		}
		
		//par here
   		DBL_INT_ADD(ctx->bitlen[0],ctx->bitlen[1],ctx->datalen * 8);
   		ctx->data[63] = ctx->bitlen[0]; 
   		ctx->data[62] = ctx->bitlen[0] >> 8; 
   		ctx->data[61] = ctx->bitlen[0] >> 16; 
   		ctx->data[60] = ctx->bitlen[0] >> 24; 
   		ctx->data[59] = ctx->bitlen[1]; 
   		ctx->data[58] = ctx->bitlen[1] >> 8; 
   		ctx->data[57] = ctx->bitlen[1] >> 16;  
   		ctx->data[56] = ctx->bitlen[1] >> 24; 
   		sha256_transform(ctx,ctx->data);	

		//we can paralized at here
   		for (i=0; i < 4; ++i) { 
      		hash[i]    = (ctx->state[0] >> (24-i*8)) & 0x000000ff; 
      		hash[i+4]  = (ctx->state[1] >> (24-i*8)) & 0x000000ff; 
      		hash[i+8]  = (ctx->state[2] >> (24-i*8)) & 0x000000ff;
      		hash[i+12] = (ctx->state[3] >> (24-i*8)) & 0x000000ff;
      		hash[i+16] = (ctx->state[4] >> (24-i*8)) & 0x000000ff;
      		hash[i+20] = (ctx->state[5] >> (24-i*8)) & 0x000000ff;
      		hash[i+24] = (ctx->state[6] >> (24-i*8)) & 0x000000ff;
      		hash[i+28] = (ctx->state[7] >> (24-i*8)) & 0x000000ff;
   		}	
		round --;
	

	}
}


__device__ int my_strcmp(uchar *str_a, uchar *str_b, uint len){
	for(int i=0; i<len; i++){
		if(str_a[i] != str_b[i])
			return false;
	}
	
	return true;
}


__global__ void sha256_wrap(uchar *pwarray, uchar *target, int* pwcount, bool *result){
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	uchar* data = (uchar*)malloc(pwcount[idx]*sizeof(uchar));
	//uchar* test = (uchar*)malloc(5*sizeof(uchar));
	//test[0] ='h';test[1]='u';test[2]='s';test[3]='k';test[4]='y';

	SHA256_CTX *ctx = new SHA256_CTX;	
	uchar hash[32];
	int round = 10000, count = 0;	

	for(int i=0; i<idx; i++){
		count += pwcount[i];
	}
	memcpy(data,&pwarray[count],pwcount[idx]*sizeof(uchar));

	sha256_hash(ctx,data,hash,pwcount[idx],round);

	//uchar* local_target = (uchar*)malloc(32*5*sizeof(uchar));
	//memcpy(local_target,target,32*5*sizeof(uchar));
	for (int i=0; i<5;i++){
		//uchar *temp =(uchar*)malloc(32*sizeof(uchar));
		//memcpy(temp,&local_target[32*i],32*sizeof(uchar));
		if(my_strcmp(hash,&target[32*i],32)){
			result[i] = true;
			memcpy(answer,data,pwcount[idx]*sizeof(uchar));
		}	
	}
}


//====================================================================
void print_hash(unsigned char hash[]){

   int idx;
   for (idx=0; idx < 32; idx++)
      printf("%02x",hash[idx]);
   printf("\n");
}

void read_hash(char filename[],unsigned char target[5][32]){
	FILE *rhash = fopen(filename,"r+");
	fread(target[0],1,32,rhash);
	fread(target[1],1,32,rhash);
	fread(target[2],1,32,rhash);
	fread(target[3],1,32,rhash);
	fread(target[4],1,32,rhash);
	fclose(rhash);
}


//============================================================================
int main(int argc, char **argv){

	std::string password;
	std::vector<std::string> pwarray;
	int dict_size = 0;
	int *pw_count;
	uchar target[5][32];
	bool result[5]; 
	
	//variable for GPU
	bool *dev_result;
	uchar *dev_target;
	uchar *dev_password;
	int *dev_pwcount;
	hipEvent_t start, stop;


	if(argc < 2){
		std::cerr << "need load dictionary!! \n";
		return 1;
	}

//timing program
	//start = clock();
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
//set up round number
	


//init result
	for(int i=0;i<5;i++){
		result[i] = false;
	}
	hipMalloc((void**)&dev_result,5*sizeof(bool));
	hipMemcpy(dev_result,result,5*sizeof(bool),hipMemcpyHostToDevice);


//read target hash
	read_hash(argv[2],target);
	//copy hash into cuda (maybe into constant memory?)
	hipMalloc((void**)&dev_target,32*5*sizeof(uchar));
	for(int i=0;i<5;i++){
		hipMemcpy((void*)&dev_target[32*i],target[i],32*sizeof(uchar),hipMemcpyHostToDevice);
	}


//read from dictionary
	std::ifstream dict(argv[1]);
	if(!dict){
		std::cerr << "No such file!! \n";
		return 1;
	}
	while(std::getline(dict, password)){
		pwarray.push_back(password);
		dict_size++;
	}

//devpassword
	pw_count = (int*)malloc(dict_size*sizeof(int));
	int temp_count = 0;
	uchar *pwstring = (uchar*)malloc(dict_size*32*sizeof(uchar));
	for(int i=0; i<dict_size; i++){
		pw_count[i] = pwarray.at(i).length();	
		//uchar *temp = new uchar[pw_count[i]];
		strcpy((char*)&pwstring[temp_count],pwarray.at(i).c_str());
		temp_count += pw_count[i];
	}
	hipMalloc((void**)&dev_password,32*dict_size*sizeof(uchar));
	hipMemcpy((void*)dev_password,pwstring,32*dict_size*sizeof(uchar),hipMemcpyHostToDevice);
	
	hipMalloc((void**)&dev_pwcount,dict_size*sizeof(int));
	hipMemcpy((void*)dev_pwcount,pw_count,dict_size*sizeof(int),hipMemcpyHostToDevice);


/*
//new malloc dev_password
	uchar** support = (uchar**)malloc(dict_size*sizeof(uchar*));
	for(int i=0; i<dict_size; i++){
		int pwlength = pwarray.at(i).length();
		uchar *temp = new uchar[pwlength];
		strcpy((char*)temp,pwarray.at(i).c_str());
		hipMalloc((void**)&support[i],32*sizeof(uchar));
		hipMemcpy(support,temp,32*sizeof(uchar),hipMemcpyHostToDevice);
	}
	
	hipMalloc((void***)&dev_password,dict_size*sizeof(uchar*));
	hipMemcpy(dev_password,support,dict_size*sizeof(uchar*), hipMemcpyDeviceToDevice);

/*
//malloc dev_password
	uchar **support;// = (uchar**)malloc(dict_size*sizeof(uchar*));
	//dev_password = (uchar**)malloc(dict_size*sizeof(uchar*));
	hipMalloc((void***)&support,dict_size*sizeof(uchar *));
	hipMemcpy((void**)dev_password,support,dict_size*sizeof(uchar *),hipMemcpyDeviceToDevice);

	//parallel here
	for(int i=0; i<dict_size; i++){
		uchar *temp = new uchar[pwarray.at(i).length()];
		strcpy((char*)temp,pwarray.at(i).c_str());
		hipMalloc((void**)&dev_password[i],99*sizeof(uchar));
		hipMemcpy((void*)dev_password[i],temp,99*sizeof(uchar),hipMemcpyHostToDevice);
	}
*/
/*	uchar *test = (uchar*)malloc(5*sizeof(uchar));
	test[0] ='h';test[1]='u';test[2]='s';test[3]='k';test[4]='y';
	uchar *dev_test;
	hipMalloc((void**)&dev_test,4*sizeof(uchar));
	hipMemcpy(dev_test,test,4*sizeof(uchar),hipMemcpyHostToDevice);
*/

	dim3 DimBlock(1024,1);
	dim3 DimGrid(20,1);
	sha256_wrap <<< DimGrid, DimBlock >>> (dev_password, dev_target, dev_pwcount, dev_result);
	hipDeviceSynchronize();

	hipEventRecord(stop);	

	hipMemcpy(result,dev_result,5*sizeof(bool),hipMemcpyDeviceToHost);

	for(int i=0; i<5; i++){
		if(result[i]){
			uchar *print = (uchar*)malloc(32*sizeof(uchar));
			hipMemcpyFromSymbol(print,HIP_SYMBOL(answer),32*sizeof(uchar));
			printf("password: %s",print);	
		}
	}
	std::cout<<std::endl;

	float time = 0;
	hipEventElapsedTime(&time,start,stop);
	std::cout<<"Total time: "<<time<<" ms"<<std::endl;

	hipFree(dev_result); hipFree(dev_target); hipFree(dev_password);
	return 0;
}
